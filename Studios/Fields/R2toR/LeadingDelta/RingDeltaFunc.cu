#include "hip/hip_runtime.h"


#include "RingDeltaFunc.h"
#include "Math/Function/NumericFunction.h"
#include "Math/Function/R2toR/Model/R2toRNumericFunction.h"

struct IsRingDeltaDomain
{


    __host__ __device__
    bool operator()(const thrust::tuple<int, int>& t)
    {
        return (thrust::get<0>(t) % 2) == 0;
    }
};

struct RingThetaGPU
{
    typedef Slab::Real argument_type;
    typedef Slab::Real result_type;

    const double a, t;
    const double rMin, dx;
    const int N;
    const double *data;

    RingThetaGPU(double a, double t, double rMin, double dx, double N, double *data)
            : a(a)
            , t(t)
            , rMin(rMin)
            , dx(dx)
            , N(N)
            , data(data)
    {           }

    __device__ Slab::Real operator()(int idx) {
        double x = rMin + (idx % N) * dx;
        double y = rMin + (idx / N) * dx;

        double r = sqrt(x*x + y*y);

        if(r-t > -dx) return a;

        return data[idx];
    }
};

struct RingDeltaGPU
{
    typedef Slab::Real argument_type;
    typedef Slab::Real result_type;

    const double eps, a, a_eps, eps_1, t;
    const double rMin, step;
    const int N;
    const double *data;

    RingDeltaGPU(double a, double eps, double t, double rMin, double step, double N, double *data)
    : eps(eps)
    , a(a)
    , a_eps(a/eps)
    , eps_1(1./eps)
    , t(t)
    , rMin(rMin)
    , step(step)
    , N(N)
    , data(data)
    {           }

    __device__ Slab::Real operator()(int idx) {
        double x = rMin + (idx % N) * step;
        double y = rMin + (idx / N) * step;

        double r = sqrt(x*x + y*y);
        double absarg = abs(r-t);

        // if(absarg < eps) return a_eps * (1-absarg*eps_1);
        if(absarg < eps) return a * delta(absarg, eps);

        return data[idx];
    }
};


bool Studios::Fields::R2toRLeadingDelta::RingDeltaFunc::renderToNumericFunction(
        Slab::Math::Base::NumericFunction<Real2D, Real> *toFunc) const {
    auto &func = *dynamic_cast<Slab::Math::R2toR::NumericFunction*>(toFunc);

    auto &outputSpace = toFunc->getSpace();
    fix &meta_data = outputSpace.getMetaData();

    const auto N = meta_data.getN(0);
    const auto h = outputSpace.getMetaData().geth(0);
    const auto xMin = func.getDomain().xMin;

    thrust::counting_iterator<int> sequence_begin(0);
    thrust::counting_iterator<int> sequence_end((int)(N * N));

    DeviceVector &deviceData = outputSpace.getDeviceData();

    auto data = thrust::raw_pointer_cast(deviceData.data());
    if(asTheta)
        thrust::transform(sequence_begin,
                          sequence_end,
                          deviceData.begin(),
                          RingThetaGPU(a, radius, xMin, h, N, data));
    else
        thrust::transform(sequence_begin,
                          sequence_end,
                          deviceData.begin(),
                          RingDeltaGPU(a, eps, radius, xMin, h, N, data));

    return true;
}