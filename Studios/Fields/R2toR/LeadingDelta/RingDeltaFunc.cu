#include "hip/hip_runtime.h"


#include "RingDeltaFunc.h"
#include "Phys/Function/DiscreteFunction.h"
#include "Allocator.h"


struct IsRingDeltaDomain
{
    __host__ __device__
    bool operator()(const thrust::tuple<int, int>& t)
    {
        return (thrust::get<0>(t) % 2) == 0;
    }
};


struct RingThetaGPU
{
    typedef Real argument_type;
    typedef Real result_type;

    const double a, t;
    const double rMin, dx;
    const int N;
    const double *data;

    RingThetaGPU(double a, double t, double rMin, double dx, double N, double *data)
    : a(a)
    , t(t)
    , rMin(rMin)
    , dx(dx)
    , N(N)
    , data(data)
    {           }

    __device__ Real operator()(int idx) {
        double x = rMin + (idx % N) * dx;
        double y = rMin + (idx / N) * dx;

        double r = sqrt(x*x + y*y);

        if(r-t >= -dx) return a;

        return data[idx];
    }
};


struct RingDeltaGPU
{
    typedef Real argument_type;
    typedef Real result_type;

    const double eps, a, t;
    const double rMin, dx;
    const int N;
    const double *data;

    RingDeltaGPU(double a, double eps, double t, double rMin, double dx, double N, double *data)
    : eps(eps)
    , a(a)
    , t(t)
    , rMin(rMin)
    , dx(dx)
    , N(N)
    , data(data)
    {           }

    __device__ Real operator()(int idx) {
        double x = rMin + (idx % N) * dx;
        double y = rMin + (idx / N) * dx;

        double r = sqrt(x*x + y*y);
        double absarg = abs(r-t);

        // if(absarg < eps) return a_eps * (1-absarg*eps_1);
        if(absarg < eps) return a * delta(absarg, eps);

        return data[idx];
    }
};


bool R2toR::LeadingDelta::RingDeltaFunc
::renderToDiscreteFunction(Base::DiscreteFunction<Real2D, Real> *toFunc) const {
    auto &outputSpace = toFunc->getSpace();
    const auto N = outputSpace.getDim().getN(0);
    const auto h = outputSpace.geth();
    const auto xMin = Numerics::Allocator::GetInstance().getNumericParams().getxLeft();

    thrust::counting_iterator<int> sequence_begin(0);
    thrust::counting_iterator<int> sequence_end(N * N);

    DeviceVector &deviceData = outputSpace.getDeviceData();

    auto data = thrust::raw_pointer_cast(deviceData.data());
    // auto func = RingDeltaGPU(a, eps, radius, xMin, h, N, data);
    auto func = RingThetaGPU(a, radius, xMin, h, N, data);

    thrust::transform(sequence_begin, sequence_end, deviceData.begin(), func);

    //thrust::tran

    return true;
}