#include "hip/hip_runtime.h"


#include "RingDeltaFunc.h"
#include "Math/Function/NumericFunction.h"
#include "Math/Function/R2toR/Model/R2toRNumericFunction.h"

struct IsRingDeltaDomain
{


    __host__ __device__
    bool operator()(const thrust::tuple<int, int>& t)
    {
        return (thrust::get<0>(t) % 2) == 0;
    }
};

struct RingThetaGPU
{
    typedef Real argument_type;
    typedef Real result_type;

    const double a, t;
    const double rMin, dx;
    const int N;
    const double *data;

    RingThetaGPU(double a, double t, double rMin, double dx, double N, double *data)
            : a(a)
            , t(t)
            , rMin(rMin)
            , dx(dx)
            , N(N)
            , data(data)
    {           }

    __device__ Real operator()(int idx) {
        double x = rMin + (idx % N) * dx;
        double y = rMin + (idx / N) * dx;

        double r = sqrt(x*x + y*y);

        if(r-t > -dx) return a;

        return data[idx];
    }
};

struct RingDeltaGPU
{
    typedef Real argument_type;
    typedef Real result_type;

    const double eps, a, a_eps, eps_1, t;
    const double rMin, step;
    const int N;
    const double *data;

    RingDeltaGPU(double a, double eps, double t, double rMin, double step, double N, double *data)
    : eps(eps)
    , a(a)
    , a_eps(a/eps)
    , eps_1(1./eps)
    , t(t)
    , rMin(rMin)
    , step(step)
    , N(N)
    , data(data)
    {           }

    __device__ Real operator()(int idx) {
        double x = rMin + (idx % N) * step;
        double y = rMin + (idx / N) * step;

        double r = sqrt(x*x + y*y);
        double absarg = abs(r-t);

        // if(absarg < eps) return a_eps * (1-absarg*eps_1);
        if(absarg < eps) return a * delta(absarg, eps);

        return data[idx];
    }
};


bool R2toR::LeadingDelta::RingDeltaFunc::renderToNumericFunction(Core::NumericFunction<Real2D, Real> *toFunc) const {
    auto &func = *dynamic_cast<R2toR::NumericFunction*>(toFunc);

    auto &outputSpace = toFunc->getSpace();
    const auto N = outputSpace.getDim().getN(0);
    const auto h = outputSpace.geth();
    const auto xMin = func.getDomain().xMin;

    thrust::counting_iterator<int> sequence_begin(0);
    thrust::counting_iterator<int> sequence_end(N * N);

    DeviceVector &deviceData = outputSpace.getDeviceData();

    auto data = thrust::raw_pointer_cast(deviceData.data());
    if(asTheta)
        thrust::transform(sequence_begin,
                          sequence_end,
                          deviceData.begin(),
                          RingThetaGPU(a, radius, xMin, h, N, data));
    else
        thrust::transform(sequence_begin,
                          sequence_end,
                          deviceData.begin(),
                          RingDeltaGPU(a, eps, radius, xMin, h, N, data));

    return true;
}