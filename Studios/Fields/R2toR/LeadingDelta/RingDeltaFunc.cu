#include "hip/hip_runtime.h"


#include "RingDeltaFunc.h"
#include "Phys/Function/DiscreteFunction.h"
#include "Allocator.h"


struct IsRingDeltaDomain
{


    __host__ __device__
    bool operator()(const thrust::tuple<int, int>& t)
    {
        return (thrust::get<0>(t) % 2) == 0;
    }
};


struct RingDeltaGPU
{
    typedef Real argument_type;
    typedef Real result_type;

    const double eps, a_eps, eps_1, t;
    const double rMin, step;
    const int N;
    const double *data;

    RingDeltaGPU(double a, double eps, double t, double rMin, double step, double N, double *data)
    : eps(eps)
    , a_eps(a/eps)
    , eps_1(1./eps)
    , t(t)
    , rMin(rMin)
    , step(step)
    , N(N)
    , data(data)
    {           }

    __device__ Real operator()(int idx) {
        double x = rMin + (idx % N) * step;
        double y = rMin + (idx / N) * step;

        double r = sqrt(x*x + y*y);
        double absarg = abs(r-t);

        if(absarg < eps) return a_eps * (1-absarg*eps_1);

        return data[idx];
    }
};


bool R2toR::LeadingDelta::RingDeltaFunc::renderToDiscreteFunction(Base::DiscreteFunction<Real2D, Real> *toFunc) const {

    auto &outputSpace = toFunc->getSpace();
    const auto N = outputSpace.getDim().getN(0);
    const auto h = outputSpace.geth();
    const auto xMin = Numerics::Allocator::getInstance().getNumericParams().getxLeft();

    thrust::counting_iterator<int> sequence_begin(0);
    thrust::counting_iterator<int> sequence_end(N * N);

    DeviceVector &deviceData = outputSpace.getDeviceData();

    thrust::transform(sequence_begin, sequence_end, deviceData.begin(),
                      RingDeltaGPU(a, eps, radius, xMin, h, N, thrust::raw_pointer_cast(deviceData.data())));

    //thrust::tran

    return true;
}