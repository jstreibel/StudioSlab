#include "hip/hip_runtime.h"
#include "hamilton-gpu.h"
#include "sim/fields/field-gpu.h"

#include "sim/math/derivatives-gpu.cuh"
#include "sim/util/device-config.h"

#include <thrust/iterator/constant_iterator.h>
#include <hip/hip_runtime.h>

extern "C" hipError_t  cuda_main();

Hamilton_GPU::Hamilton_GPU(Field_Cuda *myTempField) : temp(myTempField), N(myTempField->N) {

}

Hamilton_GPU::~Hamilton_GPU()
{
    delete temp;
}

bool Hamilton_GPU::setGPU(const unsigned int dev){
    int devCount = 0;
    hipGetDeviceCount(&devCount);
    if(dev > devCount-1) return false;

    hipError_t cuErr = hipSetDevice(dev);

    if(cuErr != hipSuccess){
        std::cout << "Error setting GPU device: " << hipGetErrorString(cuErr) << std::endl;
        return false;
    }

    int currDev;
    cuErr = hipGetDevice(&currDev);
    std::cout << "GPU device set to GPU " << currDev << "." << std::endl;

    return true;
}

floatt Hamilton_GPU::V(const floatt &phi) const { return fabs(phi); }
floatt Hamilton_GPU::dVdphi(const floatt &phi) const { return phi < 0 ? -1 : 1 ;}

struct HamiltonianStepper
{
    const floatt dT;

    HamiltonianStepper(floatt _dT) : dT(_dT) {}

    __host__ __device__
    static inline floatt dVdphi(const floatt &phi){
        // TODO: usar implementacao de sign(x) da GPU.
        return phi < 0 ? -1 : (phi > 0 ? 1 : 0);
    }

    __host__ __device__
    floatt operator ()(const floatt &phi, const floatt &d2Phidx2) const {
        return dT*(-HamiltonianStepper::dVdphi(phi) + d2Phidx2);
    }
};

Field &Hamilton_GPU::dT_F(const Field &phi, Field &out, const floatt dT,
                             const size_t begin, const size_t end)
{
    (void) begin; (void) end;

    const Field_Cuda &inField = static_cast<const Field_Cuda&>(phi);
    Field_Cuda &out1D = static_cast<Field_Cuda&>(out);

    {
        thrust::transform(inField.dPhidT_d.begin(), inField.dPhidT_d.end(),
                          thrust::constant_iterator<floatt>(dT), outField.phi_d.begin(),
                          thrust::multiplies<floatt>());
    }

    {
        DeviceVector &d2dx2_v = temp->phi_d;
        d2dx2(inField.phi_d, d2dx2_v, inField.h, N);
        thrust::transform(inField.phi_d.begin(), inField.phi_d.end(),
                          d2dx2_v.begin(), outField.dPhidT_d.begin(),
                          HamiltonianStepper(dT));
    }

    return outField;
}

floatt Hamilton_GPU::E(const Field &phi) {
    /*
    const Field_CPU &field = static_cast<const Field_Cuda&>(phi);

    const floatt *_phi = &(field.phi[0]);
    const floatt *_dphidt = &(field.dPhidT[0]);

    const size_t N = field.N;

    derivs.setf(_phi);

    //double term1 = -2*V(phi0)*p.L;
    floatt term1 = 1;
    floatt term2 = 0;
    floatt term3 = 0;
    floatt term4 = 0;
    for(size_t i=0; i<N; i++){
        floatt phiX0 = _phi[i];
        const floatt dphidx0 = derivs.dfdx(i);
        const floatt dphidt0 = _dphidt[i];

        term2 += V(phiX0);
        term3 += dphidt0*dphidt0;
        term4 += dphidx0*dphidx0;
    }

    return 0.5*(term1 + derivs.geth()*(2*term2 + term3 + term4));
    */

    return 0;
}

floatt Hamilton_GPU::Hdens(const Field &phi, const floatt x) const
{
    /*
    const Field_CPU &field = static_cast<const Field_CPU&>(phi);

    const Field_CPU::FieldOutType *_phi = &(field.phi[0]);
    const Field_CPU::FieldOutType *_dphidt = &(field.dPhidT[0]);

    derivs.setf(_phi);

    const size_t i = size_t(x/field.L * floatt(field.N));
    floatt phiX0 = _phi[i];
    const floatt dphidx0 = derivs.dfdx(i);
    const floatt dphidt0 = _dphidt[i];

    floatt term1 = -2*V(0);
    floatt term2 = V(phiX0);
    floatt term3 = dphidt0*dphidt0;
    floatt term4 = dphidx0*dphidx0;

    return term1 + term2 + term3 + term4;
    */

    return 0;
}
