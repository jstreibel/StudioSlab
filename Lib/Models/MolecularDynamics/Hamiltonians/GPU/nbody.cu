#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "nbody.cuh"
#include "params.cuh"

#include <stdio.h>
#include <stdlib.h>

__constant__ DevFloat R = R_;
__constant__ DevFloat invR = 1.0 / R_;
__constant__ unsigned int N = Nparticles;
__constant__ DevFloat dt = DT;
__constant__ DevFloat dt_2 = 0.5*DT;
__constant__ DevFloat dt2_2 = 0.5*DT*DT;
__constant__ DevFloat boxSide = 2.0 * BOX_HALF_SIDE;
__constant__ DevFloat boxHalfSide = BOX_HALF_SIDE;

#if 1
#define CHECK(cudaStatus, msg) if(cudaStatus != 0) {printf("hipError_t %i: %s - %s\n", cudaStatus, hipGetErrorString(cudaStatus), msg);}
#else
#define CHECK(cudaStatus, msg)
#endif


#ifndef __syncthreads
#define __syncthreads()
#endif
#ifndef sqrtf
#define sqrtf(a) a
#endif

const size_t SIZE = Nparticles * sizeof(Real2);
Real2 *dev_r = NULL;
Real2 *dev_p = NULL;
Real2 *dev_a = NULL;

const char *getIntMethodStr()
{
#if INT_METHOD == EULER
	return "Euler";
#elif INT_METHOD == LEAPFROG
	return "Leapfrog";
#else
	return "?";
#endif
}

/* BODY BODY */
__device__ Real2 bodyBodyInteraction(Real2 pi, Real2 pj, Real2 ai)
{
	Real2 r;

	// 2 FLOPS
	r.x = pj.x - pi.x;
	r.y = pj.y - pi.y;

	// 4 FLOPS
	Real distSqr = r.x*r.x + r.y*r.y + EPS;

	// 3 FLOPS (1 sqrt, 1 inv)
	Real dist = Sqrt(distSqr);
	Real invDist = { 1.0 / (dist) };

	// 
	if (dist < R)
	{
		const DevFloat arg = 1.0 - dist*invR;
		const DevFloat argsqr = arg*arg;
		Real invr_f = invDist * argsqr*argsqr;
		ai.x -= r.x*invr_f;
		ai.y -= r.y*invr_f;
	}

	/*
	Real invDistCube = 1.0f / (dist*distSqr);
	ai.x += r.x * invDistCube;
	ai.y += r.y * invDistCube;
	*/

	return ai;
}

__global__ void calculateForces(Real2 *dev_r, Real2 *dev_accel);
__global__ void applyForces(Real2 *dev_r, Real2 *dev_p, Real2 *dev_a);

__global__ void memSet(Real2 *devPtr, DevFloat val)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	Real2 *globalPtr = (Real2*)devPtr;

	Real2 local = globalPtr[idx];

	local.x = val;
	local.y = val;

	globalPtr[idx] = local;
}

unsigned int getN() { return Nparticles; }
Real getR() { return R_; }
Real getBoxHalfSide() { return BOX_HALF_SIDE; }

__host__ void initDevice(Real2 *host_r, Real2 *host_p, 
						 Real2 *dev_r,  Real2 *dev_p, Real2 *dev_a,
						 const size_t N, const size_t P_)
{
	if (Nparticles%P_ || ((Nparticles / P_)>1024)) {
		printf("\nTidak bagus.\n");
		exit(0);
	}

	hipError_t cudaStatus;

	/*
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dtDev), &dt, 4, 0, hipMemcpyKind::hipMemcpyHostToDevice);
	CHECK(cudaStatus, "in konstant memcpy.");
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NDev), &N, 4, 0, hipMemcpyKind::hipMemcpyHostToDevice);
	CHECK(cudaStatus, "in konstant memcpy.");
	*/

	cudaStatus = hipSetDevice(1);
	CHECK(cudaStatus, "while setting device.");

	cudaStatus = hipMalloc((void**)&dev_r, SIZE);
	CHECK(cudaStatus, "on alloc dev_r.");
	cudaStatus = hipMalloc((void**)&dev_p, SIZE);
	CHECK(cudaStatus, "on alloc dev_p.");
	cudaStatus = hipMalloc((void**)&dev_a, SIZE);
	CHECK(cudaStatus, "on alloc dev_a.");

	cudaStatus = hipMemcpy(dev_r, host_r, SIZE, hipMemcpyHostToDevice);
	CHECK(cudaStatus, "on memcpy host_r --> dev_r.");
	cudaStatus = hipMemcpy(dev_p, host_p, SIZE, hipMemcpyHostToDevice);
	CHECK(cudaStatus, "on memcpy host_p --> dev_p.");

	memSet <<<Nparticles / P_, P_ >>>(dev_a, 0.0);
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t) CHECK(hipError_t, "on damping speed.");

	CHECK(hipDeviceSynchronize(), " on device sync after init memcopies.");
}

__host__ void initDevice(Real2 *host_r, Real2 *host_p)
{
	if (Nparticles%P || ((Nparticles/P)>1024)) {
		printf("\nTidak bagus.\n");
		exit(0);
	}

	hipError_t cudaStatus;

	/*
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dtDev), &dt, 4, 0, hipMemcpyKind::hipMemcpyHostToDevice);
	CHECK(cudaStatus, "in konstant memcpy.");
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(NDev), &N, 4, 0, hipMemcpyKind::hipMemcpyHostToDevice);
	CHECK(cudaStatus, "in konstant memcpy.");
	*/

	cudaStatus = hipSetDevice(1);
	CHECK(cudaStatus, "while setting device.");

	cudaStatus = hipMalloc((void**)&dev_r, SIZE);
	CHECK(cudaStatus, "on alloc dev_r.");
	cudaStatus = hipMalloc((void**)&dev_p, SIZE);
	CHECK(cudaStatus, "on alloc dev_p.");
	cudaStatus = hipMalloc((void**)&dev_a, SIZE);
	CHECK(cudaStatus, "on alloc dev_a.");

	cudaStatus = hipMemcpy(dev_r, host_r, SIZE, hipMemcpyHostToDevice);
	CHECK(cudaStatus, "on memcpy host_r --> dev_r.");
	cudaStatus = hipMemcpy(dev_p, host_p, SIZE, hipMemcpyHostToDevice);
	CHECK(cudaStatus, "on memcpy host_p --> dev_p.");

	memSet<<<Nparticles/P, P>>>(dev_a, 0.0);
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t) CHECK(hipError_t, "on damping speed.");

	CHECK(hipDeviceSynchronize(), " on device sync after init memcopies.");
}

__host__ Real2 *fetch_rFromDevice(Real2 *rHost)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(rHost, dev_r, Nparticles * sizeof(Real2), hipMemcpyDeviceToHost);
	CHECK(cudaStatus, "on fetch r from device.");

	return rHost;

}
__host__ Real2 *fetch_pFromDevice(Real2 *pHost)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(pHost, dev_p, Nparticles * sizeof(Real2), hipMemcpyDeviceToHost);
	CHECK(cudaStatus, "on fetch p from device.");

	return pHost;
}

__global__ void dampSpeed_k(Real2 *dev_p, DevFloat factor)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	Real2 *pGlobal = (Real2*)dev_p;

	Real2 pLocal = pGlobal[idx];

	pLocal.x *= factor;
	pLocal.y *= factor;

	pGlobal[idx] = pLocal;
}
__host__ void dampSpeed(DevFloat factor)
{
	const int nBlocks = Nparticles / P;
	const int nThreadsPerBlock = P;

	dampSpeed_k <<<nBlocks, nThreadsPerBlock >>> (dev_p, factor);
	hipError_t hipError_t = hipGetLastError();
	if (hipError_t) CHECK(hipError_t, "on damping speed.");
	CHECK(hipDeviceSynchronize(), " on device sync after speed damp.");
}

void step()
{
	const int nBlocks = Nparticles / P;
	const int nThreadsPerBlock = P;

	hipError_t hipError_t;

#if INT_METHOD == LEAPFROG
	hipError_t = hipMemcpy(dev_a0, dev_a1, SIZE, hipMemcpyDeviceToDevice);
	CHECK(hipError_t, "on hipMemcpy(dev_a0, dev_a1, ...)");
#endif

	calculateForces <<<nBlocks, nThreadsPerBlock, P*sizeof(Real2)>>> (dev_r, dev_a);
	hipError_t = hipGetLastError();
	if (hipError_t) CHECK(hipError_t, "on launching calculateForces.");
	CHECK(hipDeviceSynchronize(), " on device sync after calculateForces.");
	
	applyForces <<<nBlocks, nThreadsPerBlock>>> (dev_r, dev_p, dev_a);
	hipError_t = hipGetLastError();
	if(hipError_t) CHECK(hipError_t, "on launching applyForces.");
	CHECK(hipDeviceSynchronize(), " on device sync after applyForces.");
}

__device__ Real2 bodyBoxInteraction(Real2 r, Real2 a) {
	// Collide with box:
	const DevFloat k = 1000.0;
	if      (r.x >  boxHalfSide) a.x += -k*(1.0f - boxHalfSide / r.x);
	else if (r.x < -boxHalfSide) a.x +=  k*(1.0f + boxHalfSide / r.x);
	
	if      (r.y >  boxHalfSide) a.y += -k*(1.0f - boxHalfSide / r.y);
	else if (r.y < -boxHalfSide) a.y +=  k*(1.0f + boxHalfSide / r.y);

	return a;
}

__device__ Real2 externalField(Real2 r, Real2 a) {
	a.y -= 5.e-1;
	return a;
}


__device__ Real2 tileCalculation(Real2 myPosition, Real2 accel)
{
	int i;
	extern __shared__ Real2 rShared[];

	for (i = 0; i < blockDim.x; i++)
	{
		Real2 p2 = rShared[i];
		accel = bodyBodyInteraction(myPosition, p2, accel);
#if BOUNDARY == PERIODIC
		p2.y += BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.x += BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.y -= BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.y -= BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.x -= BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.x -= BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.y += BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);
		p2.y += BOX_SIDE;
		accel = bodyBodyInteraction(myPosition, p2, accel);		
#endif

	}

	return accel;
}

__global__ void calculateForces(Real2 *dev_r, Real2 *dev_accel)
{
	extern __shared__ Real2 rShared[];

	Real2 *rGlobal = (Real2 *)dev_r;
	Real2 *accelGlobal = (Real2 *)dev_accel;
	Real2 rLocal;
	int i, tile;
	Real2 accel = {0.0f, 0.0f};
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;

	rLocal = rGlobal[gtid];

#if BOUNDARY == RIGID
	accel = bodyBoxInteraction(rLocal, accel);
#endif
#if EXTERNAL_FIELD
	accel = externalField(rLocal, accel);
#endif
	for (i = 0, tile = 0; i < N; i += P, tile++)
	{
		int idx = tile * blockDim.x + threadIdx.x;
		rShared[threadIdx.x] = rGlobal[idx];
		__syncthreads();
		accel = tileCalculation(rLocal, accel);
		__syncthreads();
	}
	// save result in global memory for the integration step.
	Real2 accel2 = {accel.x, accel.y};
	accelGlobal[gtid] = accel2;
}

__global__ void applyForces(Real2 *dev_r, Real2 *dev_p, Real2 *dev_a)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	Real2 *rGlobal = (Real2*)dev_r;
	Real2 *pGlobal = (Real2*)dev_p;
	Real2 *aGlobal = (Real2*)dev_a;

	Real2 rLocal = rGlobal[idx];
	Real2 pLocal = pGlobal[idx];
	Real2 aLocal = aGlobal[idx];

	pLocal.x += (aLocal.x)*dt_2;
	pLocal.y += (aLocal.y)*dt_2;

	rLocal.x += pLocal.x*dt;
	rLocal.y += pLocal.y*dt;

#if BOUNDARY == PERIODIC
	if (rLocal.x >  boxHalfSide) rLocal.x -= boxSide;
	else if (rLocal.x < -boxHalfSide) rLocal.x += boxSide;
	if (rLocal.y >  boxHalfSide) rLocal.y -= boxSide;
	else if (rLocal.y < -boxHalfSide) rLocal.y += boxSide;
#endif

	rGlobal[idx] = rLocal;
	pGlobal[idx] = pLocal;
}
